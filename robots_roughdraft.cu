#include "hip/hip_runtime.h"
/*

robots.cu
Written by Spencer B Liberto
12 Dec 2012

An implementation of the simulator described in this paper:
http://stephane.magnenat.net/data/Evolutionary%20Conditions%20for%20the%20Emergence%20of%20Communication%20in%20Robots%20-%20Dario%20Floreano,%20Sara%20Mitri,%20St%C3%A9phane%20Magnenat,%20Laurent%20Keller%20-%20Current%20Biology%20-%202007.pdf

*/

#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>

#define NUMCLANS 10
#define ROBOTSPERCLAN 10
#define ARENASIDELENGTH 300
#define NUMGENERATIONS 100
#define VISION 100
#define FOODXY 100
#define POISINXY 200
#define FEEDINGDISTANCE 25
#define ROBOTRADIUS 5
#define NUMGENERATIONS 100
#define NUMCYCLESPERGEN 500
#define DATALOGFILENAME "datalog.txt"

typedef struct linkedListNode
{
	linkedListNode* prev;
	linkedListNode* next;
	int correspondingRobot;
} edibleQueueNode;

typedef struct anotherLinkedListNode
{
	anotherLinkedListNode* next;
	int correspondingRobot;
	int score;
} scoreListNode;

int main(){
	runSimulation();
	return 0;
}

/* ================================== */
/* == THE CYCLES OF THE SIMULATION == */
/* ================================== */

/* runs the entire simulation from start to finish */
void runSimulation(){
	int numRobots = NUMCLANS * ROBOTSPERCLAN;
	int statesLength = numRobots * 5;
	int phenotypesLength = numRobots * 240;

	int* prevPhenotypes = malloc(numPhenotypeBits * sizeof(int));
	int* prevStates = malloc(statesLength * sizeof(int));
	int* finalScores = malloc(numRobots * sizeof(int));

	int* devicePrevPhen = 0;
	int* devicePrevStat = 0;
	int* deviceFinalScores = 0;
	hipMalloc((void**)devicePrevPhen, (phenotypesLength * sizeof(int)));
	hipMalloc((void**)devicePrevStat, (statesLength * sizeof(int)));
	hipMalloc((void**)deviceFinalScores, (numRobots * sizeof(int)));

	randomPhenotypes(prevPhenotypes);

	int i;
	for(i=0; i<NUMGENERATIONS; i++){
		randomStates(prevStates);
		hipMemcpy(devicePrevPhen, prevPhenotypes, phenotypesLength, hipMemcpyHostToDevice);
		hipMemcpy(devicePrevStat, prevStates, statesLength, hipMemcpyHostToDevice);
		generationCycle<<NUMCLANS,NUMROBOTS>>(devicePrevPhen, devicePrevStat, deviceFinalScores);
		hipMemcpy(finalScores, deviceFinalScores, numRobots, hipMemcpyDeviceToHost);
		produceNewGeneration(prevPhenotypes, finalScores, numRobots, i);
	}

	free(prevPhenotypes);
	free(prevStates);
	hipFree(devicePrevPhen);
	hipFree(devicePrevStat);
}

/* executes one generarion */
__global__ void generationCycle(int* globalPrevPhen, int* globalPrevStat, int* globalFinalScores){
	
	int tid = threadId.x;
	int bid = blockId.x;
	__shared__ int sharedPrevPhenotypes[blockPhenotypesLength];
	__shared__ int sharedPrevStates[blockPhenotypesLength];
	__shared__ edibleQueueNode edibleQueues[4]; /* [foodStart, foodEnd, PoisStart, PoisEnd] */
	__shared__ int finalScores[ROBOTSPERCLAN];

	if(tid == 0){
		int blockPhenotypesLength = ROBOTSPERCLAN * 240;
		int blockStatesLength = ROBOTSPERCLAN * 5;
		int* globalPrevBlockPhenotypes = (blockPhenotypesLength * blockIdx.x) + globalPrevPhen;
		int* globalPrevBlockStates = (blockStatesLength * blockIdx.x) + globalPrevStat;
		hipMemcpy(sharedPrevPhenotypes, globalPrevBlockPhenotypes, blockPhenotypesLength, hipMemcpyDeviceToDevice);
		hipMemcpy(sharedPrevStates, globalPrevBlockStates, blockStatesLength, hipMemcpyDeviceToDevice);
	}
	__syncthreads();

	int inputs[10];
	int outputs[3];
	int states[5];
	int phenotype[240];
	int translatedPhenotype[30];
	hipMemcpy(states, sharedPrevStates, 5, hipMemcpyDeviceToDevice);
	hipMemcpy(phenotype, sharedPrevPhenotypes, 240, hipMemcpyDeviceToDevice);
	translatePhenotype(phenotype, translatedPhenotype);

	int i;
	for(i=0; i<NUMCYCLESPERGEN; i++)
		ioCycle(tid, states, sharedPrevStates, inputs, outputs, translatedPhenotype, edibleQueues);
	for(i=0; i<ROBOTSPERCLAN; i++)
		finalScores[i] = sharedPrevStates[(5*i)+4];
	if(tid=0)
		hipMemcpy(globalFinalScores[(bid*ROBOTSPERCLAN)], finalScores, ROBOTSPERCLAN, hipMemcpyDeviceToDevice);
	__syncthreads();
}

/* executes one iocycle of a generation */
__device__ void ioCycle(int tid, int states[5], int* sharedPrevStates, int inputs[10], int outputs[3], int translatedPhenotype[30], edibleQueueNode* edibleQueues){
	/* One IOCycle */
	int bstates[5];
	int i;
	for(i=0; i<ROBOTSPERCLAN; i++){
		if (i != tid){
			hipMemcpy(bstates, (sharedPrevStates+(5*i)), 5, hipMemcpyDeviceToDevice);
			updateInputsRobots(states, bstates, inputs);
		}
	}
	updateInputsEdibles(states, inputs, edibleQueues, tid);
	updateOutputs(outputs, inputs, translatedPhenotype, states);
	updateStatesXYOB(states, outputs);
	if(tid == 0){
		tallyPoints(sharedPrevStates, edibleQueues)	
	}
	__syncthreads();
}

/* ========================================== */
/* == PRE-GENERATION RANDOM INITIAL VALUES == */
/* ========================================== */

/* produces a list of random initial states */
void randomStates(int* initialStates){
	int numRobots = NUMCLANS * ROBOTSPERCLAN;
	int i;
	for(i=0; i<numRobots; i++){
		&(initialStates++) = random(ARENASIDELENGTH);
		&(initialStates++) = random(ARENASIDELENGTH);
		&(initialStates++) = random(8) * pi / 4;
		&(initialStates++) = random(11) / 10;
		&(initialStates++) = 0;
	}
}

/* produces a list of random binart phenotypes */
void randomPhenotypes(int* initialPhenotypes){
	int numPhenotypeBits = NUMCLANS * ROBOTSPERCLAN * 480;
	int i;
	for(i=0; i<totalPhenotypeBits; i++)
		&(initialPhenotypes++) = random(2);
}

/* ======================================== */
/* == POST-GENERATION MATING AND LOGGING == */
/* ======================================== */

/* finds top 20% scorers, mates them, and produces a new generation of phenotypes. Also prints to log. */
void produceNewGeneration(int* oldPhenotypes, int* finalScores, int numRobots, int* states, int generationNumber){
	int numMedalists = floor(numRobots/5);

	/* rank scores */
	scoreListNode* scoreList = (scoreListNode *) malloc(sizeof(scoreListNode));
	scoreList->next = NULL;
	scoreList->correspondingRobot = 0;
	scoreList->score = finalScores[0];
	scoreListNode* iterScoreList;
	/* For each score, traverse list from top until lower score found */
	int i, j;
	for(i=1; i<numRobots; i++){
		scoreListNode* newNode = (scoreListNode *) malloc(sizeof(scoreListNode));
		newNode->correspondingRobot = i;
		newNode->score = finalScores[i];
		iterScoreList = scoreList;
		j=0
		while((j<i) && (finalScores[i] < iterScoreList->score)){
			iterScoreList = iterScoreList->next;
		}
		newNode->next = iterScoreList->next;
		iterScoreList->next = newNode;
	}

	/*  Output data */
	printScoreList(scoreList, numRobots, oldPhenotypes, states, generationNumber);

	/* find top scorers ("medalists") */
	int medalists[numMedalists];
	iterScoreList = scoreList;
	for(i=0; i<numMedalists; i++){
		medalist[i] = iterScoreList->correspondingRobot;
		iterScoreList = iterScoreList->next;
	}

	/* mate top scorers */
	int* deviceOldPhen;
	int* deviceNewPhen;
	int* deviceMedalists;
	hipMalloc((void**)deviceOldPhen, (numRobots * 240 * sizeof(int)));
	hipMalloc((void**)deviceNewPhen, (numRobots * 240 * sizeof(int)));
	hipMalloc((void**)deviceMedalists, (numRobots * sizeof(int)));
	hipMemcpy(deviceOldPhen, oldPhenotypes, (240*numRobots), hipMemcpyHostToDevice);
	hipMemcpy(deviceMedalists, medalists, numMedalists, hipMemcpyHostToDevice);
	mateRobotsRandomly<<1,numRobots>>(deviceOldPhen, deviceNewPhen, medalists, numMedalists);
	hipMemcpy(oldPhenotypes, deviceNewPhen, (240*numRobots), hipMemcpyDeviceToHost);
}

/* randomly mates specified robots */
__global__ void mateRobotsRandomly(int* oldPhen, int* newPhen, int* medalists, int numMedalists){

	/* Get information about parents */
	int tid = threadIdx.x;
	int* child = malloc(240*sizeof(int));
	int parenta = 0;
	int parentb = 0;
	while(parenta==parentb){
		parenta = medalist + (random(numMedalists));
		parentb = medalist + (random(numMedalists));
	}
	int* iterA = (oldPhen + 240*parenta);
	int* iterB = (oldPhen + 240*parentb);
	int* iterChild = child;
	hipMalloc((void**)iterA, (numRobots * 240 * sizeof(int)));
	hipMalloc((void**)iterB, (numRobots * 240 * sizeof(int)));
	hipMemcpy(iterA, (oldPhen + (240*parenta)), 240, hipMemcpyDeviceToDevice);
	hipMemcpy(iterB, (oldPhen + (240*parentb)), 240, hipMemcpyDeviceToDevice);

	/* Populate child */
	int i, currentA;
	for(i=0; i<240; i++){
		currentA = &iterA;
		if((&(iterA++)==(&(iterB++))) || (tid%2 == 0))
			&(child++) = currentA;
		else
			&(child++) = 1 - currentA;
	}

	/* Send child to global */
	hipMemcpy((newPhen+(240*tid)), child, 240, hipMemcpyDeviceToDevice);
}

/*  Outputs data about the current generation */
void printScoreList(scoreListNode* scoreList, int numRobots, int* phenotypes, int* states, int generationNumber){

	FILE *fp;
	fp=fopen(DATALOGFILENAME, "a");

	int translatedPhen[30];
	int currentRobot;
	int* iterPhen;
	int* iterStates;
	int i,j,k;
	fprintf(fp, "generation %d\n\nDNA\nx-coordinate y-coordinate orientation brightness final-score\n\n", generationNumber);
	for(i=0; i<numRobots; i++){
		currentRobot = scoreList->correspondingRobot;
		iterPhen = phenotypes + (240*currentRobot);
		iterStates = states + (5*currentRobot);
		for(j=0; j<30; j++){
			for(k=0; k<8; k++){
				fprintf(fp,"%d", &iterPhen);
				iterPhen++
			}
			fprintf(fp, " ");
		}
		printf("\n");
		for(j=0; j<4; j++){
			fprintf(fp,"%d   ", &(iterStates++));
		}
		pfrintf(fp,"%d\n", (scoreList->score);
		scoreList = scoreList->next;
	}

	fclose(fp);
}

/* ================================= */
/* == END-OF-CYCLE POINT COUNTING == */
/* ================================= */

/* adds or removes a robot from an edible-queue */
__device__ void queueChanging(int task, int robotInQuestion, edibleQueueNode* edibleQueues){

	switch(task){
		/* add to foodQueue */
		case 0:
			edibleQueueNode* newNode;
			hipMalloc((void**)newNode, sizeof(edibleQueueNode));
			newNode->prev = edibleQueues[1];
			newNode->next = NULL;
			newNode->correspondingRobot = robotInQuestion;
			(edibleQueues[1])->next = newNode;
			edibleQueues[1] = newNode;
			break;
		/* remove from foodQueue */
		case 1:
			edibleQueueNode* currentPointer = edibleQueues[0];
			if(currentPointer != NULL){
				while((currentPointer->correspondingRobot != robotInQuestion) || (currentPointer->next != NULL)){
					currentPointer = currentPointer->next;
				}
				if(currentPointer->correspondingRobot == robotInQuestion){
					if(currentPointer->next != NULL)
							(currentPointer->next)->prev = currentPointer->prev;
					(currentPointer->prev)->next = currentPointer->next;
				}
			}
			break;
		/* add to foodQueue */
		case 2:
			edibleQueueNode* newNode;
			hipMalloc((void**)newNode, sizeof(edibleQueueNode));
			newNode->prev = edibleQueues[3];
			newNode->next = NULL;
			newNode->correspondingRobot = robotInQuestion;
			(edibleQueues[3])->next = newNode;
			edibleQueues[3] = newNode;
			break;
		/* remove from foodQueue */
		case 3:
			edibleQueueNode* currentPointer = edibleQueues[1];
			if(currentPointer != NULL){
				while((currentPointer->correspondingRobot != robotInQuestion) || (currentPointer->next != NULL)){
					currentPointer = currentPointer->next;
				}
				if(currentPointer->correspondingRobot == robotInQuestion){
					if(currentPointer->next != NULL)
							(currentPointer->next)->prev = currentPointer->prev;
					(currentPointer->prev)->next = currentPointer->next;
				}
			}
			break;
		default:
			fprintf(stderr, "Error: queueChanging recieved an unknown task.");
	}
}

/* tallies the points after an iocycle */
__device__ void tallyPoints(int* states, edibleQueueNode* edibleQueues){
	edibleQueueNode* currentPointer = edibleQueues[0];
	
	int i;
	for(i=0; i<5; i++){
		if(currentPointer!=NULL)
			states[(((currentPointer->correspondingRobot)*5)+4)] += 1;
		else
			break;
	}

	edibleQueueNode* currentPointer = edibleQueues[2];
	for(i=0; i<5; i++){
		if(currentPointer!=NULL)
			states[(((currentPointer->correspondingRobot)*5)+4)] -= 1;
		else
			break;
	}
}

/* ========================== */
/* == IOCYCLE CALCULATIONS == */
/* ========================== */

/* updates outputs for one robot, based upon inputs and phenotype */
__device__ void updateOutputs(int outputs[3], int inputs[10], int phenotype[30], int states[5]){
	for(iterOutputs=0; iterOutputs<3; iterOutputs++){
		for(iterInputs=0; iterInputs<10; iterInputs++)
			outputs[iterOutputs] += tanh(inputs[iterInputs] * phenotype[(10 * iterOutputs) + iterInputs]);
		outputs[iterOutputs] = tanh(outputs[iterOutputs]);
	}
	states[3] = outputs;
}

/* updates inputs for one robot, based upon one other robots outputs */
__device__ void updateInputsRobots(int states[5], int bstates[5], int inputs[10]){
	int xdiff, ydiff, distance, inputBrightness, theta, bucket;

	/* differences in x and y axis */
	xdiff = states[0] - bstates[0];
	ydiff = states[1] - bstates[1];

	/* Calculate percieved brightness, break if <0 */
	distance = sqrt(pow(xdiff,2) + pow(ydiff,2));
	inputBrightness = states[3] - (distance/VISION);
	if inputBrightness<0 {
		break;
		printf("Break Error: updateInputsRelative\n")
	}

	/* calculate which bucket the light will be picked up by */
	if xdiff==0{
		if ydiff>0
			theta = pi/2;
		else 
			theta = 3*pi/2;	
	} else {
		theta = atan(ydiff/xdiff);
		if xdiff<0
			theta += pi;
	}
	bucket = floor(((states[2]+theta) *4 /pi) %8);

	/* add light value to appropriate bucket */
	inputs[bucket] += inputBrightness;
}

/* updates inputs, based upon edibles outputs */
__device__ void updateInputsEdibles(int states[5], int inputs[10], edibleQueueNode* edibleQueues, int robotInQuestion){
	int xdiff, ydiff, distance, inputBrightness, theta, bucket;

	/* CALCULATE INPUTS FROM FOOD */

	xdiff = states[0] - FOODXY;
	ydiff = states[1] - FOODXY;
	distance = sqrt(pow(xdiff,2) + pow(ydiff,2));

	/* check if on food */
	if((distance<=FEEDINGDISTANCE) && (states[8]!=1))
		queueChanging(0, robotInQuestion, edibleQueues);
	else {
		if(states[8]==1)
			queueChanging(1, robotInQuestion, edibleQueues);
		states[8] *= 0.95;
	}

	/* calculate percieved brightness from food */
	inputBrightness = 1 - (distance/(2*ARENASIDELENGTH));

	/* calculate bucket which will recieve the light input */
	if xdiff==0{
		if ydiff>0
			theta = pi/2;
		else
			theta = 3*pi/2;
	} else {
		theta = atan(ydiff/xdiff);
		if xdiff<0
			theta += pi; 
	}
	bucket = ((states[2]+theta) *4 /pi) %8;

	/* Subtract input value from appropriate light bucket */
	inputs[bucket] -= inputBrightness;

	/* CALCULATE INPUTS FROM POISIN */

	xdiff = states[0] - POISINXY;
	ydiff = states[1] - POISINXY;
	distance = sqrt(pow(xdiff,2) + pow(ydiff,2));

	/* check if on poisin */
	if((distance<=FEEDINGDISTANCE) && (states[9]!=1))
		queueChanging(2, robotInQuestion, edibleQueues);
	else {
		if(states[9]==1)
			queueChanging(3, robotInQuestion, edibleQueues);
		states[9] *= 0.95;
	}

	/* calculate percieved brightness from poisin */
	inputBrightness = 1 - (distance/(2*ARENASIDELENGTH));

	if xdiff==0{
		if ydiff>0
			theta = pi/2;
		else
			theta = 3*pi/2;
	} else {
		theta = atan(ydiff/xdiff);
		if xdiff<0
			theta+=pi;
	}
	bucket = floor(((states[2]+theta) *4 /pi) %8);

	/* Subtract input value from appropriate light bucket */
	inputs[bucket] -= inputBrightness;
}

/* translates a binary phenotype into decimal */
__device__ void translatePhenotype(int untranslated[240], int translated[30]){
	int* untransIter = untranslated[0];
	int* transIter = translated[0];

	int i, j;
	for(i=0; i<30; i++){
		*transIter = 0;
		for(j=0; j<8; j++){
			if(*(untransIter++))
				*transIter += pow(2, (7-j));
		}
		transIter++;
	}
}

/* updates the states of a robot, based upon its outputs */
__device__ void updateStatesXYOB(int states[5], int outputs[3]){
	int arcRadius, dir, relativeAngle, m, n, xmove, ymove, newx, newy, newo;

	/* Calculate new x and y coordinates
		This is a quick fix, I would love an algorithm written by
		someone who actually knows trigonometry */
	if (outputs[0] == outputs[1]) {
		if outputs[0] == 0{
			break;
			printf("Break Error: updateStatesXYO");
		}
		arcRadius = pi * ROBOTRADIUS;
		relativeAngle = 0;
	} else {
		if (outputs[0] > outputs[1]) {
			m = outputs[0];
			n = outputs[1];
			dir = 0;
		} else {
			m = outputs[1];
			n = outputs[0];
			dir = 1;
		}
		relativeAngle = (m-n)*pi;
		if n==0
			arcRadius = ROBOTRADIUS;
		else
			arcRadius = ROBOTRADIUS*m/n;
	}	
	absoluteAngle = (relativeAngle + states[2]) % (2*pi);
	if (dir)
		absoluteAngle *= -1;
	xmove = sin(absoluteAngle) * arcRadius;
	ymove = (cos(absoluteAngle) - 1) * arcRadius;
	newx = xmove + states[0];
	newy = ymove + states[1];

	/* Robots shouldn't be allowed out of the boundaries.
		This boundry-collision algorithm is not very physics-like,
		I would love an algorithm that could be more accurate */
	if (newx <= 0)
		newx = 0;
	if (newx >= ARENASIDELENGTH)
		newx = ARENASIDELENGTH;
	if (newy <= 0)
		newy = 0;
	if (newy >= ARENASIDELENGTH)
		newy = ARENASIDELENGTH;

	/* Calculate new orientation */
	if (dir)

		calc = (pi*2);

		newo = (absoluteAngle - (pi/2)) % calc;
	else
		newo = (absoluteAngle + (pi/2)) % calc;

	/* Assign new values */
	states[0] = newx;
	states[1] = newy;
	states[2] = newo;
	states[3] = outputs[2]; /* The brightness can directly transfer */
}